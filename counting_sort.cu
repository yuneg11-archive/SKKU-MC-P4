#include <hip/hip_runtime.h>

#define THREAD_NUM 1024

// Tuned for NVIDIA Tesla V100 (12 GB VRAM)
#define ARR_SPLIT_LEN 805306368 // 3 GB (= 805,306,368 * 4 Byte)

__global__ void build_histogram_kernel(int arr[], int histogram[], int arr_size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < arr_size) {
        atomicAdd(&histogram[arr[idx]], 1);
    }
}

__global__ void build_output_kernel(int prefix[], int arr[], int prefix_size, int first_prefix, int base_idx) {
    __shared__ int local_prefix[THREAD_NUM + 1];
    unsigned int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int local_idx = threadIdx.x + 1;

    if (global_idx == 0) {
        local_prefix[0] = first_prefix;
    } else if (local_idx == 1) {
        local_prefix[0] = prefix[global_idx - 1];
    }

    if (global_idx < prefix_size) {
        local_prefix[local_idx] = prefix[global_idx];

        __syncthreads();

        int cnt = local_prefix[local_idx] - local_prefix[local_idx - 1];
        int start_idx = local_prefix[local_idx - 1];

        for (int i = 0; i < cnt; i++) {
            arr[start_idx + i] = global_idx + base_idx;
        }
    }
}

__host__ void counting_sort(int arr[], int size, int max_val) {
    // Histogram
    int *arr_device;
    size_t arr_len[3] = { 0, 0, 0 };
    int arr_len_cnt = 0;
    if (size > 2 * ARR_SPLIT_LEN) {
        arr_len_cnt = 3;
        arr_len[0] = arr_len[1] = ARR_SPLIT_LEN;
        arr_len[2] = size - 2 * ARR_SPLIT_LEN;
    } else if (size > ARR_SPLIT_LEN) {
        arr_len_cnt = 2;
        arr_len[0] = ARR_SPLIT_LEN;
        arr_len[1] = size - ARR_SPLIT_LEN;
    } else {
        arr_len_cnt = 1;
        arr_len[0] = size;
    }

    int *histogram = new int[max_val];
    int *histogram_device;
    size_t histogram_len = max_val;

    hipMalloc(&arr_device, arr_len[0] * sizeof(int));
    hipMalloc(&histogram_device, histogram_len * sizeof(int));
    hipMemset(histogram_device, 0, histogram_len * sizeof(int));

    for (int i = 0; i < arr_len_cnt; i++) {
        int block_num = (arr_len[i] / THREAD_NUM) + (arr_len[i] % THREAD_NUM == 0 ? 0 : 1);
        hipMemcpy(arr_device, &arr[i * ARR_SPLIT_LEN], arr_len[i] * sizeof(int), hipMemcpyHostToDevice);
        build_histogram_kernel <<< block_num, THREAD_NUM >>> (arr_device, histogram_device, arr_len[i]);
    }

    hipMemcpy(histogram, histogram_device, histogram_len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr_device);
    hipFree(histogram_device);

    // Prefix sum
    int sum = 0;
    for (int i = 0; i < histogram_len; i++) {
        sum += histogram[i];
        histogram[i] = sum;
    }

    // Output
    int *prefix_device;
    size_t prefix_len[3] = { 0, 0, 0 };
    int prefix_len_cnt = 0;
    if (max_val > 2 * ARR_SPLIT_LEN) {
        prefix_len_cnt = 3;
        prefix_len[0] = prefix_len[1] = ARR_SPLIT_LEN;
        prefix_len[2] = max_val - 2 * ARR_SPLIT_LEN;
    } else if (max_val > ARR_SPLIT_LEN) {
        prefix_len_cnt = 2;
        prefix_len[0] = ARR_SPLIT_LEN;
        prefix_len[1] = max_val - ARR_SPLIT_LEN;
    } else {
        prefix_len_cnt = 1;
        prefix_len[0] = max_val;
    }

    int *arr_out_device;
    int arr_out_len = size;

    hipMalloc(&arr_out_device, arr_out_len * sizeof(int));
    hipMalloc(&prefix_device, prefix_len[0] * sizeof(int));

    for (int i = 0; i < prefix_len_cnt; i++) {
        int block_num = (prefix_len[i] / THREAD_NUM) + (prefix_len[i] % THREAD_NUM == 0 ? 0 : 1);
        int first_prefix = (i == 0 ? 0 : histogram[i * ARR_SPLIT_LEN - 1]);
        hipMemcpy(prefix_device, &histogram[i * ARR_SPLIT_LEN], prefix_len[i] * sizeof(int), hipMemcpyHostToDevice);
        build_output_kernel <<< block_num, THREAD_NUM >>> (prefix_device, arr_out_device, prefix_len[i], first_prefix, i * ARR_SPLIT_LEN);
    }

    hipMemcpy(arr, arr_out_device, arr_out_len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr_out_device);
    hipFree(prefix_device);

    delete [] histogram;
}
